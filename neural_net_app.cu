#include "hip/hip_runtime.h"
#ifndef NEURAL_NET_APP_H
#define NEURAL_NET_APP_H

#include "neural_net_types.h"
#include "neural_net_constants.h"
#include "neural_net_functions.h"
// #include "neural_net_app_kernel.cu"
#include "neural_net_app_kernel.h"
extern REAL *d_sunspots;



void NormalizeSunspots()
{
    REAL Min, Max;

    // Calculate Min and Max
    Min = MAX_REAL;
    Max = MIN_REAL;
    for (INT Year = 0; Year < NUM_YEARS; Year++) {
        Min = MIN(Min, Sunspots[Year]);
        Max = MAX(Max, Sunspots[Year]);
    }

    // Call the CUDA function for normalization
    normalizeSunspotsLaunch(d_sunspots, Min, Max, NUM_YEARS);

}


void InitializeApplication(NET* Net) {
    Net->Alpha = 0.5;
    Net->Eta   = 0.05;
    Net->Gain  = 1;

    REAL *d_Sunspots, *d_TrainError, *d_TestError;

    // Allocate memory and copy data to GPU
    hipMalloc(&d_Sunspots, NUM_YEARS * sizeof(REAL));
    hipMemcpy(d_Sunspots, Sunspots, NUM_YEARS * sizeof(REAL), hipMemcpyHostToDevice);

    hipMalloc(&d_TrainError, sizeof(REAL));
    hipMalloc(&d_TestError, sizeof(REAL));
    hipMemset(d_TrainError, 0, sizeof(REAL));
    hipMemset(d_TestError, 0, sizeof(REAL));

    // Calculate the number of threads and blocks for training and testing
    int threadsPerBlock = 256; 
    int totalThreads = max((TRAIN_UPB - TRAIN_LWB + 1), (TEST_UPB - TEST_LWB + 1)) * M;
    int blocks = (totalThreads + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the kernel
    CalculateError<<<blocks, threadsPerBlock>>>(d_Sunspots, Mean, d_TrainError, d_TestError, M, TRAIN_LWB, TRAIN_UPB, TEST_LWB, TEST_UPB);
    hipDeviceSynchronize();

    // Copy the results back
    hipMemcpy(&TrainErrorPredictingMean, d_TrainError, sizeof(REAL), hipMemcpyDeviceToHost);
    hipMemcpy(&TestErrorPredictingMean, d_TestError, sizeof(REAL), hipMemcpyDeviceToHost);


    // Free GPU memory
    hipFree(d_Sunspots);
    hipFree(d_TrainError);
    hipFree(d_TestError);

    // Output the results to a file
    f = fopen("BPN.txt", "w");
}



void FinalizeApplication(NET* Net)
{
  fclose(f);
}

#endif /* NEURAL_NET_APP_H */